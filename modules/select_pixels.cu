#include "hip/hip_runtime.h"
#include "../core/utility/include/Rectangle.h"

extern "C" {
__device__
bool someRectangleContainsPoint(const lightdb::Rectangle *rectangles,
        const unsigned int rectangle_count,
        const int im_x,
        const int im_y) {
    for (unsigned int i = 0; i < rectangle_count; ++i) {
        const lightdb::Rectangle &rect = rectangles[i];
        if (rect.x <= im_x
               && rect.y <= im_y
               && rect.x + rect.width >= im_x
               && rect.y + rect.height >= im_y) {
            return true;
        }
    }
    return false;
}

__global__
void select_pixels(
        unsigned char *nv12output,
        const unsigned int height,
        const unsigned int width,
        const unsigned int pitch,
        const lightdb::Rectangle *rectangles,
        const unsigned int rectangle_count,
        const unsigned int xOffset,
        const unsigned int yOffset) {
    const int im_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int im_y = blockDim.y * blockIdx.y + threadIdx.y;

    const int global_im_x = im_x + xOffset;
    const int global_im_y = im_y + yOffset;

    if (im_x < width && im_y < height) {
        const unsigned int output_luma_offset = im_x + im_y * pitch;
        const unsigned int output_luma_size = height * pitch;
        const unsigned int output_chroma_offset = output_luma_size + im_x + (im_y / 2) * pitch;

        if (!someRectangleContainsPoint(rectangles, rectangle_count, global_im_x, global_im_y)) {
            nv12output[output_luma_offset] = 0;
            nv12output[output_chroma_offset] = 128;
        }
    }
}
} // extern "C"