
#include <hip/hip_runtime.h>
extern "C"
__global__
void blur(unsigned char* input, unsigned char* output, const unsigned int width, const unsigned int height) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x = index % width;
    int y = (index-x)/width;
    int size = 3;

    if(index < width * height) {
        unsigned int output_red = 0,
                     output_green = 0,
                     output_blue = 0;

        int applications = 0;
        for(int delta_x = -size; delta_x < size+1; ++delta_x) {
            for(int delta_y = -size; delta_y < size+1; ++delta_y) {
                if(x + delta_x >= 0 && x + delta_x < width && y + delta_y >= 0 && y + delta_y < height) {
                    const int currentIndex = (index+delta_x+delta_y*width)*3;
                    output_red += input[currentIndex];
                    output_green += input[currentIndex+1];
                    output_blue += input[currentIndex+2];
                    applications++;
                }
            }
        }

        output[index*3] = output_red / applications;
        output[index*3+1] = output_green / applications;
        output[index*3+2] = output_blue / applications;
    }
}

extern "C"
__global__
void overlay(unsigned char* left, unsigned char* right, unsigned char* output,
             const unsigned int width, const unsigned int height,
             const unsigned int transparent_color) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(right[index] != transparent_color)
        left[index] = right[index];
}
