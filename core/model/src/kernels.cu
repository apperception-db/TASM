
#include <hip/hip_runtime.h>
extern "C"
__global__
void blur(unsigned char* input, unsigned char* output, const unsigned int width, const unsigned int height) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x = index % width;
    int y = (index-x)/width;
    int size = 3;

    if(index < width * height) {
        unsigned int output_red = 0,
                     output_green = 0,
                     output_blue = 0;

        int applications = 0;
        for(int delta_x = -size; delta_x < size+1; ++delta_x) {
            for(int delta_y = -size; delta_y < size+1; ++delta_y) {
                if(x + delta_x >= 0 && x + delta_x < width && y + delta_y >= 0 && y + delta_y < height) {
                    const int currentIndex = (index+delta_x+delta_y*width)*3;
                    output_red += input[currentIndex];
                    output_green += input[currentIndex+1];
                    output_blue += input[currentIndex+2];
                    applications++;
                }
            }
        }

        output[index*3] = output_red / applications;
        output[index*3+1] = output_green / applications;
        output[index*3+2] = output_blue / applications;
    }
}

extern "C"
__global__
void overlay(unsigned char* left, unsigned char* right, unsigned char* output,
             const unsigned int width, const unsigned int height,
             const unsigned int transparent_color) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(right[index] != transparent_color)
        left[index] = right[index];
}

extern "C"
__global__
void resize(unsigned char* input, unsigned char* output,
            const unsigned int input_width, const unsigned int input_height,
            const unsigned int output_width, const unsigned int output_height,
            const float fx, const float fy) {
    const unsigned int output_x = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int output_y = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int input_index = output_y * input_width + output_x;

    if (output_x < output_width && output_y < output_height)
    {
        const float src_x = output_x * fx;
        const float src_y = output_y * fy;
        const unsigned int output_index = src_y * output_width + src_x;

        output[output_index] = input[input_index];
    }
}
