#include "hip/hip_runtime.h"
#include "stdio.h"

extern "C"
__global__
void blur(unsigned char* input, unsigned char* output, const unsigned int width, const unsigned int height) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x = index % width;
    int y = (index-x)/width;
    int size = 3;

    if(index < width * height) {
        unsigned int output_red = 0,
                     output_green = 0,
                     output_blue = 0;

        int applications = 0;
        for(int delta_x = -size; delta_x < size+1; ++delta_x) {
            for(int delta_y = -size; delta_y < size+1; ++delta_y) {
                if(x + delta_x >= 0 && x + delta_x < width && y + delta_y >= 0 && y + delta_y < height) {
                    const int currentIndex = (index+delta_x+delta_y*width)*3;
                    output_red += input[currentIndex];
                    output_green += input[currentIndex+1];
                    output_blue += input[currentIndex+2];
                    applications++;
                }
            }
        }

        output[index*3] = output_red / applications;
        output[index*3+1] = output_green / applications;
        output[index*3+2] = output_blue / applications;
    }
}

extern "C"
__global__
void overlay(unsigned char* left, unsigned char* right, unsigned char* output,
             const unsigned int width, const unsigned int height,
             const unsigned int transparent_color) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(right[index] != transparent_color)
        left[index] = right[index];
}

extern "C"
__global__
void resize(unsigned char* input, unsigned char* output,
            const unsigned int input_width, const unsigned int input_height,
            const unsigned int output_width, const unsigned int output_height,
            const float fx, const float fy) {
    const unsigned int output_x = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int output_y = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int input_index = output_y * input_width + output_x;

    if (output_x < output_width && output_y < output_height)
    {
        const float src_x = output_x * fx;
        const float src_y = output_y * fy;
        const unsigned int output_index = src_y * output_width + src_x;

        output[output_index] = input[input_index];
    }
}

#define COLOR_COMPONENT_BIT_SIZE 10
#define COLOR_COMPONENT_MASK     0x3FF
// From OpenCV https://github.com/opencv/opencv/blob/master/modules/cudacodec/src/cuda/nv12_to_rgb.cu

extern "C"
__constant__ float constHueColorSpaceMat[9] = {1.1644f, 0.0f, 1.596f,
                                               1.1644f, -0.3918f, -0.813f,
                                               1.1644f, 2.0172f, 0.0f};

extern "C"
__device__ static void YUV2RGB(const unsigned int* yuvi, float* red, float* green, float* blue)
{
    float luma, chromaCb, chromaCr;

    // Prepare for hue adjustment
    luma     = (float)yuvi[0];
    chromaCb = (float)((int)yuvi[1] - 512.0f);
    chromaCr = (float)((int)yuvi[2] - 512.0f);

    // Convert YUV To RGB with hue adjustment
    red[0]   = (luma     * 1.1644f) +
             (chromaCb * 0.0f) +
             (chromaCr * 1.596f);

    green[0] = (luma     * 1.1644f) +
             (chromaCb * -0.3918f) +
             (chromaCr * -0.813f);

    blue[0]  = (luma     * 1.1644f) +
             (chromaCb * 2.0172f) +
             (chromaCr * 0.0f);
}

extern "C"
__device__ static unsigned int RGBA_pack_10bit(float red, float green, float blue, unsigned int alpha)
{
    unsigned int ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = ::fmin(::fmax(red,   0.0f), 1023.f);
    green = ::fmin(::fmax(green, 0.0f), 1023.f);
    blue  = ::fmin(::fmax(blue,  0.0f), 1023.f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = (((unsigned int)blue  >> 2) |
      (((unsigned int)green >> 2) << 8)  |
      (((unsigned int)red   >> 2) << 16) |
      (unsigned int)alpha);

    return ARGBpixel;
}

extern "C"
__global__ void NV12_to_RGB(const unsigned char* srcImage, const unsigned int nSourcePitch,
                            unsigned int* dstImage, const unsigned int nDestPitch,
                            unsigned int width, unsigned int height)
{
    //const unsigned int nSourcePitch = 4096;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    const int y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width || y >= height)
        return;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values

    unsigned int yuv101010Pel[2];

    yuv101010Pel[0] = (srcImage[y * nSourcePitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImage[y * nSourcePitch + x + 1]) << 2;

    const unsigned int chromaOffset = nSourcePitch * height;

    const int y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        unsigned int chromaCb = srcImage[chromaOffset + y_chroma * nSourcePitch + x    ];
        unsigned int chromaCr = srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImage[chromaOffset + (y_chroma + 1) * nSourcePitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImage[chromaOffset + (y_chroma + 1) * nSourcePitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
        yuv101010Pel[0] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    // this steps performs the color conversion
    unsigned int yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] =  (yuv101010Pel[0] &   COLOR_COMPONENT_MASK    );
    yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] =  (yuv101010Pel[1] &   COLOR_COMPONENT_MASK    );
    yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA

    const unsigned int dstImagePitch = nDestPitch >> 2;

    dstImage[y * dstImagePitch + x     ] = RGBA_pack_10bit(red[0], green[0], blue[0], ((unsigned int)0xff << 24));
    dstImage[y * dstImagePitch + x + 1 ] = RGBA_pack_10bit(red[1], green[1], blue[1], ((unsigned int)0xff << 24));
}