#include "hip/hip_runtime.h"
#include "stdio.h"

extern "C"
__global__
void blur(unsigned char* input, unsigned char* output, const unsigned int width, const unsigned int height) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x = index % width;
    int y = (index-x)/width;
    int size = 3;

    if(index < width * height) {
        unsigned int output_red = 0,
                     output_green = 0,
                     output_blue = 0;

        int applications = 0;
        for(int delta_x = -size; delta_x < size+1; ++delta_x) {
            for(int delta_y = -size; delta_y < size+1; ++delta_y) {
                if(x + delta_x >= 0 && x + delta_x < width && y + delta_y >= 0 && y + delta_y < height) {
                    const int currentIndex = (index+delta_x+delta_y*width)*3;
                    output_red += input[currentIndex];
                    output_green += input[currentIndex+1];
                    output_blue += input[currentIndex+2];
                    applications++;
                }
            }
        }

        output[index*3] = output_red / applications;
        output[index*3+1] = output_green / applications;
        output[index*3+2] = output_blue / applications;
    }
}

extern "C"
__global__
void overlay(unsigned char* left, unsigned char* right, unsigned char* output,
             const unsigned int width, const unsigned int height,
             const unsigned int transparent_color) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(right[index] != transparent_color)
        left[index] = right[index];
}

extern "C"
__global__
void resize(const unsigned int* input, unsigned int* output,
            const unsigned int input_width, const unsigned int input_height,
            const unsigned int output_width, const unsigned int output_height,
            const float fx, const float fy) {
    const unsigned int output_x = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int output_y = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int output_index = output_y * output_width + output_x;

    if (output_x < output_width && output_y < output_height)
    {
        const unsigned int src_x = output_x / fx;
        const unsigned int src_y = output_y / fy;
        const unsigned int input_index = src_y * input_width + src_x;

        output[output_index] = input[input_index];
    }
}

extern "C"
__global__
void resize_weighted(
            const unsigned int* input, float* output,
            const unsigned int input_width, const unsigned int input_height,
            const unsigned int output_width, const unsigned int output_height,
            const float fx, const float fy) {
    const unsigned int output_x = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int output_y = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int output_index = output_y * output_width + output_x;

    if (output_x < output_width && output_y < output_height)
    {
        const unsigned int src_x = output_x / fx;
        const unsigned int src_y = output_y / fy;
        const unsigned int input_index = src_y * input_width + src_x;

        const unsigned int rgba = input[input_index];
        const unsigned int channel_size = output_height * output_width;

        output[output_index] =                  (rgba & 0xff) / 255.0f;
        output[output_index + channel_size] =   ((rgba >> 8) & 0xff) / 255.0f;
        output[output_index + 2*channel_size] = ((rgba >> 16) & 0xff) / 255.0f;
    }
}

/*extern "C"
__global__
void extract_luma_and_resize(
            const unsigned int* input, unsigned int* output,
            const unsigned int input_width, const unsigned int input_height,
            const unsigned int output_width, const unsigned int output_height,
            const float fx, const float fy) {
    const unsigned int output_x = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int output_y = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int output_index = output_y * output_width + output_x;

    if (output_x < output_width && output_y < output_height)
    {
        const unsigned int src_x = output_x / fx;
        const unsigned int src_y = output_y / fy;
        const unsigned int input_index = src_y * input_width + src_x;

        output[output_index] = input[input_index];
    }
}*/

typedef struct {
    int classes;
    char **names;
} metadata;
typedef struct {
    float x, y, w, h;
} box;


extern "C"
__global__
void draw_detections(
            unsigned char* nv12image,
            const unsigned int pitch,
            const unsigned int width,
            const unsigned int height,
            const float *probabilities,
            const unsigned int class_count,
            const box *boxes,
            const unsigned int box_count,
            const float fx, const float fy) {
    const unsigned int prediction_class = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int box_id = blockDim.y * blockIdx.y + threadIdx.y;

    const unsigned int prediction_index = box_id * class_count + prediction_class;

    if(prediction_class < class_count &&
            box_id < box_count &&
            probabilities[prediction_index] > 0.001)
    {

        const box &b = boxes[box_id];

        const unsigned int x1 = (b.x - b.w/2) / fx;
        const unsigned int x2 = (b.x + b.w/2) / fx;
        const unsigned int y1 = (b.y - b.h/2) / fy;
        const unsigned int y2 = (b.y + b.h/2) / fy;
        const unsigned int image_x1 = min(x1, width - 3);
        const unsigned int image_x2 = min(x2, width - 3);
        const unsigned int image_y1 = min(y1, height - 3);
        const unsigned int image_y2 = min(y2, height - 3);

        nv12image[image_y1 * pitch + image_x1] = 0;
        nv12image[image_y2 * pitch + image_x2] = 0;
        nv12image[image_y1 * pitch + image_x1] = 0;
        nv12image[image_y2 * pitch + image_x2] = 0;

/*        for(int y = image_y1; y < image_y2; y++)
        {
            nv12image[y * pitch + image_x1] = 0;
            nv12image[(y+1) * pitch + image_x1] = 0;
            nv12image[(y+2) * pitch + image_x1] = 0;
        }
        for(int y = image_y1; y < image_y2; y++)
        {
            nv12image[y * pitch + image_x2] = 0;
            nv12image[(y+1) * pitch + image_x2 + 1] = 0;
            nv12image[(y+2) * pitch + image_x2 + 2] = 0;
        }
        for(int x = image_x1; x < image_x2; x++)
        {
            nv12image[image_y1 * pitch + x] = 0;
            nv12image[image_y1 * pitch + x + 1] = 0;
            nv12image[image_y1 * pitch + x + 2] = 0;
        }
        for(int x = image_x1; x < image_x2; x++)
        {
            nv12image[image_y2 * pitch + x] = 0;
            nv12image[image_y2 * pitch + x + 1] = 0;
            nv12image[image_y2 * pitch + x + 2] = 0;
        }*/
    }
}

#define COLOR_COMPONENT_BIT_SIZE 10
#define COLOR_COMPONENT_MASK     0x3FF
// From OpenCV https://github.com/opencv/opencv/blob/master/modules/cudacodec/src/cuda/nv12_to_rgb.cu

extern "C"
__constant__ float constHueColorSpaceMat[9] = {1.1644f, 0.0f, 1.596f,
                                               1.1644f, -0.3918f, -0.813f,
                                               1.1644f, 2.0172f, 0.0f};

extern "C"
__device__ static void YUV2RGB(const unsigned int* yuvi, float* red, float* green, float* blue)
{
    float luma, chromaCb, chromaCr;

    // Prepare for hue adjustment
    luma     = (float)yuvi[0];
    chromaCb = (float)((int)yuvi[1] - 512.0f);
    chromaCr = (float)((int)yuvi[2] - 512.0f);

    // Convert YUV To RGB with hue adjustment
    red[0]   = (luma     * 1.1644f) +
             (chromaCb * 0.0f) +
             (chromaCr * 1.596f);

    green[0] = (luma     * 1.1644f) +
             (chromaCb * -0.3918f) +
             (chromaCr * -0.813f);

    blue[0]  = (luma     * 1.1644f) +
             (chromaCb * 2.0172f) +
             (chromaCr * 0.0f);
}

extern "C"
__device__ static unsigned int RGBA_pack_10bit(float red, float green, float blue, unsigned int alpha)
{
    unsigned int ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = ::fmin(::fmax(red,   0.0f), 1023.f);
    green = ::fmin(::fmax(green, 0.0f), 1023.f);
    blue  = ::fmin(::fmax(blue,  0.0f), 1023.f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = (((unsigned int)blue  >> 2) |
      (((unsigned int)green >> 2) << 8)  |
      (((unsigned int)red   >> 2) << 16) |
      (unsigned int)alpha);

    return ARGBpixel;
}

extern "C"
__global__ void NV12_to_RGB(const unsigned char* srcImage, const unsigned int nSourcePitch,
                            unsigned int* dstImage, const unsigned int nDestPitch,
                            unsigned int width, unsigned int height)
{
    //const unsigned int nSourcePitch = 4096;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    const int y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width || y >= height)
        return;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values

    unsigned int yuv101010Pel[2];

    yuv101010Pel[0] = (srcImage[y * nSourcePitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImage[y * nSourcePitch + x + 1]) << 2;

    const unsigned int chromaOffset = nSourcePitch * height;

    const int y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        unsigned int chromaCb = srcImage[chromaOffset + y_chroma * nSourcePitch + x    ];
        unsigned int chromaCr = srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImage[chromaOffset + (y_chroma + 1) * nSourcePitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImage[chromaOffset + (y_chroma + 1) * nSourcePitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
        yuv101010Pel[0] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x    ] << ( COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= ((unsigned int)srcImage[chromaOffset + y_chroma * nSourcePitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    // this steps performs the color conversion
    unsigned int yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] =  (yuv101010Pel[0] &   COLOR_COMPONENT_MASK    );
    yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] =  (yuv101010Pel[1] &   COLOR_COMPONENT_MASK    );
    yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA

    const unsigned int dstImagePitch = nDestPitch >> 2;

    dstImage[y * dstImagePitch + x     ] = RGBA_pack_10bit(red[0], green[0], blue[0], ((unsigned int)0xff << 24));
    dstImage[y * dstImagePitch + x + 1 ] = RGBA_pack_10bit(red[1], green[1], blue[1], ((unsigned int)0xff << 24));
}